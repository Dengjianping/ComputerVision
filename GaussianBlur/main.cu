#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>

#include "hip/hip_runtime.h"
#include ""

#include <opencv2\core.hpp>
#include <opencv2\highgui.hpp>
#include <opencv2\imgcodecs.hpp>
#include <opencv2\imgproc.hpp>

using namespace std;
using namespace cv;

__constant__ float PI = 3.1415;

__device__ float twoDimGaussian(int x, int y, float theta)
{
    float coeffient = 1 / (2 * PI*powf(theta, 2));
    float powerIndex = -(powf(x, 2) + powf(y, 2)) / (2 * powf(theta, 2));
    return coeffient*expf(powerIndex);
}

__global__ void gaussianBlur(uchar* input, size_t srcPitch, int rows, int cols, uchar* output, size_t dstPitch, int radius, float theta = 1.0)
{
    // use a 1-dim array to store gaussian matrix
    extern __shared__ float gaussian[];

    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    if (row < 2 * radius + 1 && col < 2 * radius + 1)
    {
        gaussian[row*(2 * radius + 1) + col] = twoDimGaussian(col - radius, radius - row, theta);
    }
    __syncthreads();

    if (row < rows&&col < cols)
    {
        for (size_t i = 0; i < 2 * radius + 1; i++)
        {
            for (size_t j = 0; j < 2 * radius + 1; j++)
            {
                // convolving, about how addressing matrix in device, 
                // see this link http://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html#group__CUDART__MEMORY_1g32bd7a39135594788a542ae72217775c
                uchar *inputValue = (uchar *)((char *)input + row*srcPitch) + col;
                uchar *outputValue = (uchar *)((char *)output + (row + i)*dstPitch) + (col + j);
                *outputValue += (*inputValue) * gaussian[i*(2 * radius + 1) + j];
            }
        }
    }
}

void gaussianBlur(const Mat & src, const Mat & dst, int radius, float theta = 1.0)
{
    // define blocks size and threads size
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceCount - 1);
    
    /*
    my sample image size is 600 * 450, so we need 600 * 450 threads to process this image on device at least, 
    each block can contain 1024 threads at most in my device, so ,I can define block size as 600 * 450 / 1024 = 263 (20 * 15)
    */
    int blockCount = (int)(dst.rows * dst.cols / prop.maxThreadsPerBlock) + 1;
    blockCount = (int)(sqrt(blockCount)) + 1;
    dim3 blockSize(blockCount, blockCount);
    dim3 threadSize(32, 32);
    
    // create 2 streams to asynchronously copy data to device
    hipStream_t srcStream, dstStream;
    hipStreamCreate(&srcStream); hipStreamCreate(&dstStream);

    // copy data to device
    int channelCount = src.channels();
    switch (channelCount)
    {
    // handle 1 channel image
    case 1:
        uchar* srcData; float1* dstData;
        
        size_t srcPitch;
        hipMallocPitch(&srcData, &srcPitch, sizeof(uchar)*src.cols, src.rows);
        hipMemcpy2DAsync(srcData, srcPitch, src.data, src.cols*sizeof(uchar), src.cols*sizeof(uchar), src.rows, hipMemcpyHostToDevice, srcStream);
        
        size_t dstPitch;
        hipMallocPitch(&dstData, &dstPitch, sizeof(float1)*dst.cols, dst.rows);
        hipMemcpy2DAsync(dstData, dstPitch, dst.data, dst.cols*sizeof(float1), dst.cols*sizeof(float1), dst.rows, hipMemcpyHostToDevice, dstStream);
        hipStreamSynchronize(srcStream); hipStreamSynchronize(dstStream);
        
        int dynamicSize = (2 * radius + 1)*(2 * radius + 1) * sizeof(float);
        gaussianBlur<<<blockSize, threadSize, dynamicSize>>> (srcData, srcPitch, src.rows, src.cols, dstData, dstPitch, radius);
        
        hipError_t error = hipDeviceSynchronize();
        if (error != hipSuccess)
        {
            cout << hipGetErrorString(error) << endl;
        }
        hipMemcpy(dst.data, dstData, sizeof(float1)*dst.rows*dst.cols, hipMemcpyDeviceToHost);       
        
        // recource releasing
        hipFree(srcData); hipFree(dstData);
    default:
        break;
    }
    cudaStreamdsttroy(srcStream); cudaStreamdsttroy(dstStream);
}


int main(void)
{
    string path = "type-c.jpg";
    
    // source image
    Mat hostInput = imread(path, IMREAD_GRAYSCALE);
    
    // gaussian kernel radius, the size is 2 * radius + 1, odd number is convenient for computing
    int radius = 2;
    Mat hostOutput(Size(hostInput.rows + 2*radius, hostInput.cols + 2*radius), CV_32F, Scalar(0));
    
    gaussianBlur(hostInput, hostOutput, radius)
    
    /* 
    need to convert to CV_8U type, because a CV_32F image, whose pixel value ranges from 0.0 to 1.0
    http://stackoverflow.com/questions/14539498/change-type-of-mat-object-from-cv-32f-to-cv-8u
    */
    hostOutput.convertTo(result, CV_8U);

    string title = "CUDA";
    namedWindow(title);
    imshow(title, hostOutput);

    waitKey(0);
    //system("pause");
    return 0;
}