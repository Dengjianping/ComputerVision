#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <vector>

#include "hip/hip_runtime.h"
#include ""
#include "thrust\host_vector.h"
#include "thrust\device_vector.h"

#include <opencv2\core.hpp>
#include <opencv2\highgui.hpp>
#include <opencv2\imgcodecs.hpp>
#include <opencv2\imgproc.hpp>
#include <opencv2\cudaarithm.hpp>
#include <opencv2\cudalegacy.hpp>
#include <opencv2\cudaimgproc.hpp>

using namespace std;
using namespace cv;
//using namespace thrust;

__constant__ float PI = 3.1415;

/*float twoDimGaussian(int x, int y, float theta)
{
    float coeffient = 1 / (2 * PI*pow(theta, 2));
    float powerIndex = -(pow(x, 2) + pow(y, 2)) / (2 * pow(theta, 2));
    return coeffient*exp(powerIndex);
}*/

__device__ float twoDimGaussian(int x, int y, float theta)
{
    float coeffient = 1 / (2 * PI*powf(theta, 2));
    float powerIndex = -(powf(x, 2) + powf(y, 2)) / (2 * powf(theta, 2));
    return coeffient*expf(powerIndex);
}

/*void initGaussianMatrix(thrust::host_vector<thrust::host_vector<float> >* matrix, int radius, float theta = 1.0)
{
    for (size_t i = 0; i < 2 * radius + 1; i++)
    {
        thrust::host_vector<float> t;
        for (size_t j = 0; j < 2 * radius + 1; j++)
        {
            float gaussianValue = twoDimGaussian(j - radius, radius - i, theta);
            t.push_back(gaussianValue);
        }
        matrix->push_back(t);
    }
}*/

__global__ void gaussianBlur(uchar* input, int rows, int cols, uchar* output, int radius, float theta = 1.0)
{
    extern __shared__ float gaussian[];

    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    if (row < 2 * radius + 1 && col < 2 * radius + 1)
    {
        gaussian[row*(2 * radius + 1) + col] = twoDimGaussian(col - radius, radius - row, theta);
    }
    __syncthreads();

    if (row < rows&&col < cols)
    {
        for (size_t i = 0; i < 2 * radius + 1; i++)
        {
            for (size_t j = 0; j < 2 * radius + 1; j++)
            {
                output(row + i, col + j) += (float)input(row, col) * gaussian[i*(2 * radius + 1) + j];
            }
        }
    }
}

void gaussianBluring(const Mat & src, const Mat & des, int radius, float theta = 1.0)
{
    // define blocks size and threads size
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceCount - 1);
    int blockCount = (int)(des.rows * des.cols / prop.maxThreadsPerBlock) + 1;
    blockCount = (int)(sqrt(blockCount)) + 1;
    dim3 blockSize(blockCount, blockCount);
    dim3 threadSize(32, 32);

    // copy data to device
    int channelCount = src.channels();
    switch (channelCount)
    {
    case 1:
        uchar* srcData; float1* desData;
        hipMalloc((void**)&srcData, sizeof(uchar)*src.rows*src.cols);
        hipMalloc((void**)&desData, sizeof(uchar)*des.rows*des.cols);

        hipStream_t srcStream, desStream;
        hipStreamCreate(&srcStream); hipStreamCreate(&desStream);
        hipMemcpyAsync(srcData, src.data, sizeof(uchar)*src.rows*src.cols, hipMemcpyHostToDevice, srcStream);
        hipMemcpyAsync(desData, des.data, sizeof(float1)*des.rows*des.cols, hipMemcpyHostToDevice, desStream);
        // block here until the data copy is finished
        hipStreamSynchronize(srcStream); hipStreamSynchronize(desStream);

        hipMemset(desData, 0, sizeof(float1)*des.rows*des.cols);

        //call kernel function
        int dynamicSize = (2 * radius + 1)*(2 * radius + 1) * sizeof(float);
        gaussianBlur<<<blockSize, threadSize, dynamicSize>>> ()

        // get data back to host
        hipError_t error = hipDeviceSynchronize();
        if (error != hipSuccess)
        {
            cout << hipGetErrorString(error) << endl;
        }
        hipMemcpy(des.data, desData, sizeof(float1)*des.rows*des.cols, hipMemcpyDeviceToHost);

        // recource releasing
        hipStreamDestroy(srcStream); hipStreamDestroy(desStream);
        hipFree(srcData); hipFree(desData);
    default:
        break;
    }
}


int main(void)
{
    string path = "type-c.jpg";
    
    // source image
    _InputArray hostInput = imread(path, IMREAD_GRAYSCALE);
    cuda::GpuMat deviceInput = hostInput.getGpuMat();
    cout << path << endl;
    
    // gaussian kernel radius, the size is 2 * radius + 1, odd number is convenient for computing
    int radius = 2;
    
    InputArray hostResult = Mat(Size(deviceInput.cols + 2 * radius, deviceInput.rows + 2 * radius), CV_32F, Scalar(0));
    cuda::GpuMat deviceResult = hostResult.getGpuMat();

    // so the matrix size is 2 * radius + 1, use even number is convenient for computing.
    thrust::host_vector<thrust::host_vector<float> > hostGaussianMatrix;
    //initGaussianMatrix(&hostGaussianMatrix, radius);
    thrust::device_vector<thrust::device_vector<float> > deviceGaussianMatrix = hostGaussianMatrix;
    /*
    my sample image size is 600 * 450, so we need 600 * 450 threads to process this image on device at least, 
    each block can contain 1024 threads at most in my device, so ,I can define block size as 600 * 450 / 1024 = 263 (20 * 15)
    */
    /*int deviceCount;
    hipGetDeviceCount(&deviceCount);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceCount - 1);
    //int blockCount = (int)(hostResult.rows * hostResult.cols / prop.maxThreadsPerBlock) + 1;*/
    //blockCount = (int)(sqrt(blockCount)) + 1;
    //dim3 blockSize(blockCount, blockCount);
    dim3 blockSize(17, 17);
    dim3 threadSize(32, 32);
    
    gaussianBlur <<<blockSize, threadSize>>> (deviceInput, deviceResult, deviceGaussianMatrix, radius);
    hipError_t error = hipDeviceSynchronize();
    if (error != hipSuccess)
    {
        cout << hipGetErrorString(error) << endl;
    }

    //deviceResult.download(hostResult);
    //Mat result(Size(input.cols + 2 * radius, input.rows + 2 * radius), CV_32F, Scalar(0));
    //convolutionMatrix(input, gaussianKenrel, result);
    
    /* 
    need to convert to CV_8U type, because a CV_32F image, whose pixel value ranges from 0.0 to 1.0
    http://stackoverflow.com/questions/14539498/change-type-of-mat-object-from-cv-32f-to-cv-8u
    */
    //result.convertTo(result, CV_8U);

    string title = "CUDA";
    namedWindow(title);
    imshow(title, hostInput);

    waitKey(0);
    //system("pause");
    return 0;
}