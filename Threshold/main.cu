#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>

#include <opencv2\core.hpp>
#include <opencv2\highgui.hpp>
#include <opencv2\imgproc.hpp>

#include "hip/hip_runtime.h"
#include ""

#define MAX_THREADS 32

using namespace std;
using namespace cv;

__global__ void threshold(uchar *input, size_t inputPitch, int rows, int cols, uchar *output, uchar thresholdValue) {
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    if (row < rows&&col < cols) {
        uchar *pixelValue = (uchar*)((char*)input + row*inputPitch) + col;
        uchar *outputPixelValue = (uchar*)((char*)output + row*inputPitch) + col;
        if (*pixelValue < thresholdValue) {
            *outputPixelValue = 0;
        }
        else {
            *outputPixelValue = 255;
        }
    }
}

void thresholdImage(const Mat & input, Mat & output, uchar threholdValue) {
    output = Mat(input.size(), CV_8U, Scalar(0));

    uchar *d_input, *d_output;
    size_t inputPitch, outputPitch;
    hipMallocPitch(&d_input, &inputPitch, sizeof(uchar)*input.cols, input.rows);
    hipMallocPitch(&d_output, &outputPitch, sizeof(uchar)*output.cols, output.rows);

    hipStream_t inputCopy, outputCopy;
    hipStreamCreate(&inputCopy); hipStreamCreate(&outputCopy);

    hipMemcpy2DAsync(d_input, inputPitch, input.data, sizeof(uchar)*input.cols, sizeof(uchar)*input.cols, input.rows, hipMemcpyHostToDevice, inputCopy);
    hipMemcpy2DAsync(d_output, outputPitch, output.data, sizeof(uchar)*output.cols, sizeof(uchar)*output.cols, output.rows, hipMemcpyHostToDevice, outputCopy);

    hipStreamSynchronize(inputCopy); hipStreamSynchronize(outputCopy);

    dim3 blockSize(input.cols / MAX_THREADS + 1, input.rows / MAX_THREADS + 1);
    dim3 threadSize(MAX_THREADS, MAX_THREADS);

    threshold<<<blockSize, threadSize>>> (d_input, inputPitch, input.rows, input.cols, d_output, threholdValue);

    hipError_t error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        cout << hipGetErrorString(error) << endl;
    }

    hipMemcpy2D(output.data, output.cols * sizeof(uchar), d_output, inputPitch, output.cols * sizeof(uchar), output.rows, hipMemcpyDeviceToHost);

    hipStreamDestroy(inputCopy); hipStreamDestroy(outputCopy);
    hipFree(d_input); hipFree(d_output);
}

int main() {
    string path = "type-c.jpg";
    Mat img = imread(path, IMREAD_GRAYSCALE);
    Mat img1 = imread(path);

    Mat result;

    hipEvent_t start, end;
    hipEventCreate(&start); hipEventCreate(&end);
    hipEventRecord(start);
    thresholdImage(img, result, 50);
    hipEventRecord(end);
    hipEventSynchronize(start); hipEventSynchronize(end);
    float time;
    hipEventElapsedTime(&time, start, end);
    hipEventDestroy(start); hipEventDestroy(end);
    cout << "time cost on cpu: " << time << " ms." << endl;

    Mat th;
    double cpuStart = (double)getTickCount();
    threshold(img, th, 50, 255, img.type());
    double cpuEnd = (double)getTickCount();
    double cpuTime = (cpuEnd - cpuStart) / getTickFrequency();
    cout << "time cost on cpu: " << cpuTime * 1000 << " ms." << endl;

    string title = "CUDA";
    namedWindow(title);
    imshow(title, result);
    imshow("CPU", th);
    waitKey(0);
    imwrite("threshold.jpg", result);

    return 0;
}