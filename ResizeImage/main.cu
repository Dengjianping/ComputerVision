#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>

#include "hip/hip_runtime.h"
#include ""

#include <opencv2\core.hpp>
#include <opencv2\highgui.hpp>
#include <opencv2\imgcodecs.hpp>
#include <opencv2\imgproc.hpp>

#define MAX_THREADS 32

using namespace std;
using namespace cv;

__global__ void enlarge(float* src, size_t inputPitch, int rows, int cols, float* dst, size_t outputPitch, float rowRatio, float colRatio)
{
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    float x = float(col - cols / 2);
    float y = float(rows / 2 - row);

    if (row < rows&&col < cols)
    {
        // get 4 coordinators points back
        float* q11 = (float*)((char*)src + (int)(row*rowRatio)*inputPitch) + (int)(col*colRatio);
        float* q12 = (float*)((char*)src + ((int)(row*rowRatio) + 1)*inputPitch) + (int)(col*colRatio);
        float* q21 = (float*)((char*)src + (int)(row*rowRatio)*inputPitch) + (int)(col*colRatio) + 1;
        float* q22 = (float*)((char*)src + ((int)(row*rowRatio) + 1)*inputPitch) + (int)(col*colRatio) + 1;

        // Bilinear Interpolation
        float* outputPixel = (float*)((char*)dst + row*inputPitch) + col;
        *outputPixel = (1 - rowRatio)*(1 - colRatio)*(*q11) + (1 - rowRatio)*colRatio*(*q12) + rowRatio*(1 - colRatio)*(*q21) + rowRatio*colRatio*(*q22);
    }
}

__global__ void shrink()
{}

void resizeImage(const Mat & input, Mat & output, float alpha)
{
    float rowRatio = (float)output.rows / (float)input.rows;
    float colRatio = (float)output.cols / (float)output.cols;
    // define block size and thread size
    dim3 blockSize(output.cols / MAX_THREADS + 1, output.rows / MAX_THREADS + 1);
    dim3 threadSize(MAX_THREADS, MAX_THREADS);

    hipStream_t inputStream, outputStream;
    hipStreamCreate(&inputStream); hipStreamCreate(&outputStream);

    size_t inputPitch, outputPitch;
    float* src; float* dst;
    hipMallocPitch(&src, &inputPitch, sizeof(float)*input.cols, input.rows);
    hipMemcpy2DAsync(src, inputPitch, input.data, sizeof(float)*input.cols, sizeof(float)*input.cols, input.rows, hipMemcpyHostToDevice, inputStream);

    hipMallocPitch(&dst, &outputPitch, sizeof(float)*output.cols, output.rows);
    hipMemcpy2DAsync(dst, outputPitch, output.data, sizeof(float)*output.cols, sizeof(float)*output.cols, output.rows, hipMemcpyHostToDevice, outputStream);

    hipStreamSynchronize(inputStream); hipStreamSynchronize(outputStream);

    //enlarge <<<blockSize, threadSize >>> ();
    hipError_t error = hipDeviceSynchronize();
    if (error != hipSuccess)
    {
        cout << hipGetErrorString(error) << endl;
    }
    hipMemcpy2D(output.data, sizeof(float)*output.cols, dst, outputPitch, sizeof(float)*output.cols, output.rows, hipMemcpyDeviceToHost);

    // resource releasing
    hipStreamDestroy(inputStream); hipStreamDestroy(outputStream);
    hipFree(src); hipFree(dst);
}

int main()
{
    string path = "type-c.jpg";
    Mat img = imread(path, IMREAD_GRAYSCALE);
    float alpha = 1.5;
    Mat result(Size(img.cols*alpha, img.rows*alpha), CV_8U, Scalar(0));

    string title = "CUDA";
    namedWindow(title);
    imshow(title, img);
    waitKey(0);

    return 0;
}