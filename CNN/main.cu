#include <stdio.h>
#include <iostream>
#include <string>

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "hiprand.h"

#include <opencv2\core.hpp>
#include <opencv2\highgui.hpp>
#include <opencv2\imgcodecs.hpp>
#include <opencv2\imgproc.hpp>

#define MAX_THREADS 32

using namespace std;
using namespace cv;

void randCUDA()
{
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

}

void unnanedFunction(const Mat & img)
{
    int channelCount = img.channels();
    vector<Mat> chs;
    split(img, chs);

    float* channelData1; float* channelData2; float* channelData3;
    hipStream_t channelStream1; hipStream_t channelStream2; hipStream_t channelStream3;
    size_t channelPitch1; size_t channelPitch2; size_t channelPitch3;

    // allocate channel 1
    hipMallocPitch(&channelData1, &channelPitch1, sizeof(float)*img.cols, img.row);
    hipMemcpy2DAsync(channelData1, channelPitch1, chs[0].data, sizeof(float)*img.cols, sizeof(float)*img.rows, img.rows, hipMemcpyHostToDevice, channelStream1);

    hipMallocPitch(&channelData2, &channelPitch2, sizeof(float)*img.cols, img.row);
    hipMemcpy2DAsync(channelData2, channelPitch2, chs[1].data, sizeof(float)*img.cols, sizeof(float)*img.rows, img.rows, hipMemcpyHostToDevice, channelStream2);

    hipMallocPitch(&channelData3, &channelPitch3, sizeof(float)*img.cols, img.row);
    hipMemcpy2DAsync(channelData3, channelPitch3, chs[2].data, sizeof(float)*img.cols, sizeof(float)*img.rows, img.rows, hipMemcpyHostToDevice, channelStream3);


}

int main()
{
    string path = "type-c.jpg";
    Mat img = imread(path);

    string title = "CUDA";
    namedWindow(title);
    imshow(title, img);
    waitKey(0);

    return 0;
}